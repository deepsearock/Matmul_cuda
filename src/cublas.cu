#include <iostream>
#include <cstdlib>
#include <chrono>
#include <hipblas.h>
#include <hip/hip_runtime.h>

void matrixMultiplyCUBLAS(float* A, float* B, float* C, int M, int N, int K) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1.0f;
    float beta = 0.0f;

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, M * N * sizeof(float));
    hipMalloc((void**)&d_B, N * K * sizeof(float));
    hipMalloc((void**)&d_C, M * K * sizeof(float));

    float *h_A = new float[M * K];
    float *h_B = new float[K * N];
    float *h_C = new float[M * N];
    float *h_C_ref = new float[M * N];

    //initialize host memory with random values
    for (int i = 0; i < M * K; ++i) h_A[i] = static_cast<float>(rand()) / RAND_MAX;
    for (int i = 0; i < K * N; ++i) h_B[i] = static_cast<float>(rand()) / RAND_MAX;

    //allocate and copy memory to device
    hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice);

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, K, M, N, &alpha, d_B, K, d_A, N,  &beta, d_C, K);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    //copy results back to host
    hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_C_ref;

    hipblasDestroy(handle);
}


//quick cublas implementation to check for performance
int main(int argc, char* argv[]) {

    if (argc != 5 || std::string(argv[1]) != "-i") {
        std::cerr << "Usage: " << argv[0] << " -i <rowDimA> <colDimA> <colDimB>" << std::endl;
        return 1;
    }

    int M = std::atoi(argv[2]);
    int N = std::atoi(argv[3]);
    int K = std::atoi(argv[4]);

    float* A = new float[M * N];
    float* B = new float[N * K];
    float* C = new float[M * K];

    for (int i = 0; i < M * N; ++i) A[i] = static_cast<float>(rand()) / RAND_MAX;
    for (int i = 0; i < N * K; ++i) B[i] = static_cast<float>(rand()) / RAND_MAX;

    auto start = std::chrono::high_resolution_clock::now();
    matrixMultiplyCUBLAS(A, B, C, M, N, K);
    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> duration = end - start;
    double time_sec = duration.count();
    double num_operations = 2.0 * M * N * K;
    double tflops = num_operations / (time_sec * 1e12);

    std::cout << "Matrix multiplication took " << time_sec << " seconds." << std::endl;
    std::cout << "Performance: " << tflops << " TFLOPS." << std::endl;

    delete[] A;
    delete[] B;
    delete[] C;
    
    return 0;
}
