#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Kernel for matrix multiplication using shared memory and tiling
__global__ void matrixMulShared(float *A, float *B, float *C, int M, int N, int K, int BLOCK_SIZE) {
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    int threadRow = threadIdx.y;
    int threadCol = threadIdx.x;
    
    extern __shared__ float sharedMem[];
    float *As = sharedMem;
    float *Bs = &sharedMem[BLOCK_SIZE * BLOCK_SIZE];
    
    float Cvalue = 0.0f;
    
    for (int m = 0; m < (K + BLOCK_SIZE - 1) / BLOCK_SIZE; ++m) {
        int globalArow = blockRow * BLOCK_SIZE + threadRow;
        int globalAcol = m * BLOCK_SIZE + threadCol;
        int globalBrow = m * BLOCK_SIZE + threadRow;
        int globalBcol = blockCol * BLOCK_SIZE + threadCol;
        
        // Handle boundary conditions to prevent out-of-bounds memory access
        if (globalArow < M && globalAcol < K)
            As[threadRow * BLOCK_SIZE + threadCol] = A[globalArow * K + globalAcol];
        else
            As[threadRow * BLOCK_SIZE + threadCol] = 0.0f;
        
        if (globalBrow < K && globalBcol < N)
            Bs[threadRow * BLOCK_SIZE + threadCol] = B[globalBrow * N + globalBcol];
        else
            Bs[threadRow * BLOCK_SIZE + threadCol] = 0.0f;
        
        __syncthreads();
        
        // Perform matrix multiplication for the tile
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Cvalue += As[threadRow * BLOCK_SIZE + k] * Bs[k * BLOCK_SIZE + threadCol];
        }
        
        __syncthreads();
    }
    
    int globalCrow = blockRow * BLOCK_SIZE + threadRow;
    int globalCcol = blockCol * BLOCK_SIZE + threadCol;
    
    // Ensure the output is written only within valid matrix bounds
    if (globalCrow < M && globalCcol < N) {
        C[globalCrow * N + globalCcol] = Cvalue;
    }
}

void checkCudaError(hipError_t error, const char *message) {
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error: %s: %s\n", message, hipGetErrorString(error));
        exit(-1);
    }
}

int main(int argc, char *argv[]) {
    if (argc != 5 || strcmp(argv[1], "-i") != 0) {
        fprintf(stderr, "Usage: %s -i <rowDimA> <colDimA> <colDimB>\n", argv[0]);
        return -1;
    }
    
    int M = atoi(argv[2]); // Rows of A
    int K = atoi(argv[3]); // Columns of A, rows of B
    int N = atoi(argv[4]); // Columns of B
    
    int BLOCK_SIZE = 16; // Default block size
    
    float *A = (float *)malloc(M * K * sizeof(float));
    float *B = (float *)malloc(K * N * sizeof(float));
    float *C = (float *)malloc(M * N * sizeof(float));
    
    for (int i = 0; i < M * K; i++) {
        A[i] = (float)(rand() % 100) / 100.0f;
    }
    for (int i = 0; i < K * N; i++) {
        B[i] = (float)(rand() % 100) / 100.0f;
    }
    
    printf("Running matrix multiplication with dimensions A(%d x %d), B(%d x %d)\n", M, K, K, N);
    matrixMultiply(A, B, C, M, N, K, BLOCK_SIZE);
    
    free(A);
    free(B);
    free(C);
    
    return 0;
}
