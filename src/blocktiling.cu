#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Kernel for matrix multiplication using shared memory and tiling
__global__ void matrixMulShared(float *A, float *B, float *C, int M, int N, int K, int BLOCK_SIZE) {
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    int threadRow = threadIdx.y;
    int threadCol = threadIdx.x;
    
    extern __shared__ float sharedMem[];
    float *As = sharedMem;
    float *Bs = &sharedMem[BLOCK_SIZE * BLOCK_SIZE];
    
    float Cvalue = 0.0f;
    
    for (int m = 0; m < (K + BLOCK_SIZE - 1) / BLOCK_SIZE; ++m) {
        int globalArow = blockRow * BLOCK_SIZE + threadRow;
        int globalAcol = m * BLOCK_SIZE + threadCol;
        int globalBrow = m * BLOCK_SIZE + threadRow;
        int globalBcol = blockCol * BLOCK_SIZE + threadCol;
        
        // Handle boundary conditions to prevent out-of-bounds memory access
        if (globalArow < M && globalAcol < K)
            As[threadRow * BLOCK_SIZE + threadCol] = A[globalArow * K + globalAcol];
        else
            As[threadRow * BLOCK_SIZE + threadCol] = 0.0f;
        
        if (globalBrow < K && globalBcol < N)
            Bs[threadRow * BLOCK_SIZE + threadCol] = B[globalBrow * N + globalBcol];
        else
            Bs[threadRow * BLOCK_SIZE + threadCol] = 0.0f;
        
        __syncthreads();
        
        // Perform matrix multiplication for the tile
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Cvalue += As[threadRow * BLOCK_SIZE + k] * Bs[k * BLOCK_SIZE + threadCol];
        }
        
        __syncthreads();
    }
    
    int globalCrow = blockRow * BLOCK_SIZE + threadRow;
    int globalCcol = blockCol * BLOCK_SIZE + threadCol;
    
    // Ensure the output is written only within valid matrix bounds
    if (globalCrow < M && globalCcol < N) {
        C[globalCrow * N + globalCcol] = Cvalue;
    }
}

void checkCudaError(hipError_t error, const char *message) {
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error: %s: %s\n", message, hipGetErrorString(error));
        exit(-1);
    }
}

// Host function to manage memory and invoke the kernel
double matrixMultiply(float *A, float *B, float *C, int M, int N, int K, int BLOCK_SIZE) {
    float *d_A, *d_B, *d_C;
    size_t sizeA = M * K * sizeof(float);
    size_t sizeB = K * N * sizeof(float);
    size_t sizeC = M * N * sizeof(float);
    
    checkCudaError(hipMalloc((void **)&d_A, sizeA), "hipMalloc d_A failed");
    checkCudaError(hipMalloc((void **)&d_B, sizeB), "hipMalloc d_B failed");
    checkCudaError(hipMalloc((void **)&d_C, sizeC), "hipMalloc d_C failed");
    
    checkCudaError(hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice), "hipMemcpy A->d_A failed");
    checkCudaError(hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice), "hipMemcpy B->d_B failed");
    
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
    int sharedMemSize = 2 * BLOCK_SIZE * BLOCK_SIZE * sizeof(float);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    matrixMulShared<<<dimGrid, dimBlock, sharedMemSize>>>(d_A, d_B, d_C, M, N, K, BLOCK_SIZE);
    checkCudaError(hipGetLastError(), "Kernel launch failed");
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    double ops = 2.0 * M * N * K;
    double tflops = (ops / (milliseconds / 1000.0)) / 1e12;
    
    checkCudaError(hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost), "hipMemcpy d_C->C failed");
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return tflops;
}

int main(int argc, char *argv[]) {
    if (argc != 5 || strcmp(argv[1], "-i") != 0) {
        fprintf(stderr, "Usage: %s -i <rowDimA> <colDimA> <colDimB>\n", argv[0]);
        return -1;
    }
    
    int M = atoi(argv[2]); // Rows of A
    int K = atoi(argv[3]); // Columns of A, rows of B
    int N = atoi(argv[4]); // Columns of B
    
    int BLOCK_SIZE = 32; // Default block size
    
    float *A = (float *)malloc(M * K * sizeof(float));
    float *B = (float *)malloc(K * N * sizeof(float));
    float *C = (float *)malloc(M * N * sizeof(float));
    
    for (int i = 0; i < M * K; i++) {
        A[i] = (float)(rand() % 100) / 100.0f;
    }
    for (int i = 0; i < K * N; i++) {
        B[i] = (float)(rand() % 100) / 100.0f;
    }
    
    printf("Running matrix multiplication with dimensions A(%d x %d), B(%d x %d)\n", M, K, K, N);
    double tflops = matrixMultiply(A, B, C, M, N, K, BLOCK_SIZE);
    printf("Execution Time: %f ms, Performance: %f TFLOPS\n", tflops * 1e12 / (2.0 * M * N * K), tflops);
    
    free(A);
    free(B);
    free(C);
    
    return 0;
}
