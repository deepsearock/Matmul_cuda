#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Kernel for matrix multiplication using shared memory and tiling
__global__ void matrixMulShared(float *A, float *B, float *C, int M, int N, int K, int BLOCK_SIZE) {
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    int threadRow = threadIdx.y;
    int threadCol = threadIdx.x;
    
    extern __shared__ float sharedMem[];
    float *As = sharedMem;
    float *Bs = &sharedMem[BLOCK_SIZE * BLOCK_SIZE];
    
    float Cvalue = 0.0f;
    
    for (int m = 0; m < (K + BLOCK_SIZE - 1) / BLOCK_SIZE; ++m) {
        int globalArow = blockRow * BLOCK_SIZE + threadRow;
        int globalAcol = m * BLOCK_SIZE + threadCol;
        int globalBrow = m * BLOCK_SIZE + threadRow;
        int globalBcol = blockCol * BLOCK_SIZE + threadCol;
        
        // Handle boundary conditions to prevent out-of-bounds memory access
        if (globalArow < M && globalAcol < K)
            As[threadRow * BLOCK_SIZE + threadCol] = A[globalArow * K + globalAcol];
        else
            As[threadRow * BLOCK_SIZE + threadCol] = 0.0f;
        
        if (globalBrow < K && globalBcol < N)
            Bs[threadRow * BLOCK_SIZE + threadCol] = B[globalBrow * N + globalBcol];
        else
            Bs[threadRow * BLOCK_SIZE + threadCol] = 0.0f;
        
        __syncthreads();
        
        // Perform matrix multiplication for the tile
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Cvalue += As[threadRow * BLOCK_SIZE + k] * Bs[k * BLOCK_SIZE + threadCol];
        }
        
        __syncthreads();
    }
    
    int globalCrow = blockRow * BLOCK_SIZE + threadRow;
    int globalCcol = blockCol * BLOCK_SIZE + threadCol;
    
    // Ensure the output is written only within valid matrix bounds
    if (globalCrow < M && globalCcol < N) {
        C[globalCrow * N + globalCcol] = Cvalue;
    }
}

void checkCudaError(hipError_t error, const char *message) {
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error: %s: %s\n", message, hipGetErrorString(error));
        exit(-1);
    }
}

int main(int argc, char *argv[]) {
    if (argc != 5 || strcmp(argv[1], "-i") != 0) {
        fprintf(stderr, "Usage: %s -i <rowDimA> <colDimA> <colDimB>\n", argv[0]);
        return -1;
    }
    
    int M = atoi(argv[2]); // Rows of A
    int K = atoi(argv[3]); // Columns of A, rows of B
    int N = atoi(argv[4]); // Columns of B
    
    int blockSizes[] = {8, 16, 32};
    int numBlocks = sizeof(blockSizes) / sizeof(blockSizes[0]);
    
    float *A = (float *)malloc(M * K * sizeof(float));
    float *B = (float *)malloc(K * N * sizeof(float));
    float *C = (float *)malloc(M * N * sizeof(float));
    
    for (int i = 0; i < M * K; i++) {
        A[i] = (float)(rand() % 100) / 100.0f;
    }
    for (int i = 0; i < K * N; i++) {
        B[i] = (float)(rand() % 100) / 100.0f;
    }
    
    for (int b = 0; b < numBlocks; b++) {
        int BLOCK_SIZE = blockSizes[b];
        double totalTime = 0.0;
        double totalTflops = 0.0;
        int runs = 100;
        
        for (int i = 0; i < runs; i++) {
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);
            
            double tflops = matrixMultiply(A, B, C, M, N, K, BLOCK_SIZE);
            
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            
            totalTime += milliseconds;
            totalTflops += tflops;
            
            hipEventDestroy(start);
            hipEventDestroy(stop);
        }
        
        double avgTime = totalTime / runs;
        double avgTflops = totalTflops / runs;
        
        printf("Block Size: %d, Average Execution Time: %f ms, Average Performance: %f TFLOPS\n", BLOCK_SIZE, avgTime, avgTflops);
    }
    
    free(A);
    free(B);
    free(C);
    
    return 0;
}
