#include <iostream>
#include <chrono>
#include <cstdlib>
#include <getopt.h>
#include <hip/hip_runtime.h>
#include "matrix_multiply_tiled.cuh"
#include "matrix_multiply_naive.cuh"
#include "utils.cuh"

void printUsage() {
    std::cout << "Usage: TiledMatrixMul -i <rowDimA> <colDimA> <colDimB>" << std::endl;
    std::cout << "  <rowDimA>: Number of rows in matrix A and matrix C" << std::endl;
    std::cout << "  <colDimA>: Number of columns in matrix A (and number of rows in matrix B)" << std::endl;
    std::cout << "  <colDimB>: Number of columns in matrix B and matrix C" << std::endl;
    exit(1);
}

int main(int argc, char *argv[]) {
    if (argc != 5 || std::string(argv[1]) != "-i") {
        printUsage();
    }

    int rowDimA = std::atoi(argv[2]);
    int colDimA = std::atoi(argv[3]);
    int colDimB = std::atoi(argv[4]);

    printGpuSpecs();

    std::cout << "Matrix dimensions: " << std::endl;
    std::cout << "  A (" << rowDimA << "x" << colDimA << ")" << std::endl;
    std::cout << "  B (" << colDimA << "x" << colDimB << ")" << std::endl;
    std::cout << "  C (" << rowDimA << "x" << colDimB << ")" << std::endl;

    int blockSizes[] = {8, 16, 32};
    int tileSizes[] = {8, 16, 32};

    double memoryVolumeBytes = (rowDimA * colDimA + colDimA * colDimB + rowDimA * colDimB) * sizeof(float);
    double memoryVolumeGB = memoryVolumeBytes / 1e9;

    for (int blockSize : blockSizes) {
        for (int tileSize : tileSizes) {
            double totalNaiveTime = 0.0, totalNaiveFlops = 0.0;
            double totalTiledTime = 0.0, totalTiledFlops = 0.0;

            for (int run = 0; run < 10; ++run) {
                auto naiveResult = runMatrixMulNaive(rowDimA, colDimB, colDimA, blockSize);
                totalNaiveTime += naiveResult.second;
                totalNaiveFlops += naiveResult.first;
                //double naiveMemoryBandwidth = memoryVolumeGB / (totalNaiveTime / 1000.0);

                auto tiledResult = runMatrixMulTiled(rowDimA, colDimB, colDimA, tileSize);
                totalTiledTime += tiledResult.second;
                totalTiledFlops += tiledResult.first;
                //double tiledMemoryBandwidth = memoryVolumeGB / (totalTiledTime / 1000.0);
            }

            double avgNaiveTime = totalNaiveTime / 10.0;
            double avgNaiveFlops = totalNaiveFlops / 10.0;
            double avgTiledTime = totalTiledTime / 10.0;
            double avgTiledFlops = totalTiledFlops / 10.0;

            std::cout << "\nPerformance Results:" << std::endl;
            std::cout << "Block Size: " << blockSize << ", Tile Size: " << tileSize, ", Block Size: 8x" << tileSize << std::endl;
            std::cout << "Naive Execution Time (ms): " << avgNaiveTime << std::endl;
            std::cout << "Tiled Execution Time (ms): " << avgTiledTime << std::endl;
            std::cout << "Naive Performance (TFLOPS): " << avgNaiveFlops << std::endl;
            std::cout << "Tiled Performance (TFLOPS): " << avgTiledFlops << std::endl;
            //std::cout << "Naive Memory Bandwidth (GB/s): " << naiveMemoryBandwidth << std::endl;
            //std::cout << "Tiled Memory Bandwidth (GB/s): " << tiledMemoryBandwidth << std::endl;
        }
    }
    return 0;
}
