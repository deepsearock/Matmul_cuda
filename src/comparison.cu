#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include "matrix_multiply_shared.cuh"
#include "matrix_multiply_naive.cuh"

#include <hip/hip_runtime.h>

int main(int argc, char *argv[]) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    hipGetDeviceProperties(&prop, 0);
    printf("GPU: %s, Compute Capability: %d.%d, Global Memory: %.2f GB, Shared Memory per Block: %d KB, Max Threads per Block: %d \n\n", prop.name, prop.major, prop.minor, prop.totalGlobalMem / (1024.0 * 1024.0 * 1024.0), (int)(prop.sharedMemPerBlock / 1024), (int)prop.maxThreadsPerBlock);
    
    int M = atoi(argv[2]);
    int K = atoi(argv[3]);
    int N = atoi(argv[4]);
    int blockSizes[] = {8, 16, 32};
    int numBlocks = sizeof(blockSizes) / sizeof(blockSizes[0]);
    
    float *A = (float *)malloc(M * K * sizeof(float));
    float *B = (float *)malloc(K * N * sizeof(float));
    float *C = (float *)malloc(M * N * sizeof(float));
    
    for (int i = 0; i < M * K; i++) A[i] = (float)(rand() % 100) / 100.0f;
    for (int i = 0; i < K * N; i++) B[i] = (float)(rand() % 100) / 100.0f;
    
    printf("Matrix Multiplication Performance Comparison:");
    printf("%-12s %-20s %-20s %-20s %-20s %-20s %-20s %-20s %-20s", "Block Size", "Shared TFLOPS", "Shared Time (ms)", "Naive TFLOPS", "Naive Time (ms)", "Theor. Warps", "Ach. Warps", "Theor. Occ. (%)", "Ach. Occ. (%)");
    printf("%-12s %-20s %-20s %-20s %-20s\n", "Block Size", "Shared TFLOPS", "Shared Time (ms)", "Naive TFLOPS", "Naive Time (ms)");
    
    for (int i = 0; i < numBlocks; i++) {
        int BLOCK_SIZE = blockSizes[i];
        float execTimeShared = 0.0f, execTimeNaive = 0.0f;
        double tflopsShared = matrixMultiplyShared(A, B, C, M, N, K, BLOCK_SIZE, &execTimeShared);
        double tflopsNaive = matrixMultiplyNaive(A, B, C, M, N, K, BLOCK_SIZE, &execTimeNaive);
        
        int maxThreadsPerSM;
        int warpSize;
        hipDeviceGetAttribute(&maxThreadsPerSM, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0);
        hipDeviceGetAttribute(&warpSize, hipDeviceAttributeWarpSize, 0);
        int maxWarpsPerSM = maxThreadsPerSM / warpSize;
        int theoreticalWarps = maxWarpsPerSM;
        
        int activeWarpsPerSM;
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&activeWarpsPerSM, matrixMultiplyShared, BLOCK_SIZE * BLOCK_SIZE, 0);
        double achievedWarps = (double)activeWarpsPerSM;
        
        int minGridSize;
        int blockSize;
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, matrixMultiplyShared, 0, 0);
        int maxBlocksPerSM = minGridSize;
        double theoreticalOccupancy = ((double)maxBlocksPerSM * BLOCK_SIZE * BLOCK_SIZE) / maxThreadsPerSM * 100.0;
        
        int achievedActiveThreads;
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&achievedActiveThreads, matrixMultiplyShared, BLOCK_SIZE * BLOCK_SIZE, 0);
        double achievedOccupancy = (double)achievedActiveThreads / maxBlocksPerSM * 100.0;
        
        
        
        printf("%-12d %-20.2f %-20.2f %-20.2f %-20.2f %-20d %-20.2f %-20.2f %-20.2f\n", BLOCK_SIZE, tflopsShared, execTimeShared, tflopsNaive, execTimeNaive, theoreticalWarps, achievedWarps, theoreticalOccupancy, achievedOccupancy);
    }
    
    free(A);
    free(B);
    free(C);
    
    return 0;
}
